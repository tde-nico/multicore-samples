#include "hip/hip_runtime.h"
#include "hello.h"

#define N 1000

__global__ void	sum(int *v1_d, int *v2_d, int *v3_d, int n)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	if (id < n)
		v3_d[id] = v1_d[id] + v2_d[id];
}

int	main(void)
{
	int	v1_h[N];
	int	v2_h[N];
	int	v3_h[N];

	for (int i = 0; i < N; ++i)
	{
		v1_h[i] = i + 1;
		v2_h[i] = -i;
	}

	int	*v1_d;
	int	*v2_d;
	int	*v3_d;

	hipMalloc((void **)&v1_d, N * sizeof(int));
	hipMalloc((void **)&v2_d, N * sizeof(int));
	hipMalloc((void **)&v3_d, N * sizeof(int));

	hipMemcpy(v1_d, v1_h, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(v2_d, v2_h, N * sizeof(int), hipMemcpyHostToDevice);

	sum<<<4,256>>>(v1_d, v2_d, v3_d, N);

	hipMemcpy(v3_h, v3_d, N * sizeof(int), hipMemcpyDeviceToHost);


	for (int i = 0; i < N; ++i)
	{
		if (v3_h[i] != 1)
			printf("Error: %d %d\n", i, v3_h[i]);
	}

	hipFree(v1_d);
	hipFree(v2_d);
	hipFree(v3_d);

	hipDeviceSynchronize();
	return (0);
}
