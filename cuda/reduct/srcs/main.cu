#include "hip/hip_runtime.h"
#include "hello.h"

#define N (1<<11)
//#define N (1<<16)

__global__ void	reduct(int *v, int *r)
{
	int	id = threadIdx.x + blockIdx.x * blockDim.x;
	int	stride = N / 2;

	for (int i = stride; i > 0; i >>= 1)
	{
		if (id < i)
			v[id] += v[id + i];
		__syncthreads();
	}
	if (!id)
		*r = v[0];
}

int	main(void)
{
	int	v1_h[N];
	int	res_h;
	int	*v1_d;
	int	*res_d;
	for (int i = 0; i < N; ++i)
	{
		v1_h[i] = 3;
	}


	hipMalloc((void **)&v1_d, N * sizeof(int));
	hipMalloc((void **)&res_d, sizeof(int));

	hipMemcpy(v1_d, v1_h, N * sizeof(int), hipMemcpyHostToDevice);

	reduct<<< 1, 1024 >>>(v1_d, res_d);
	//reduct<<< 32, 1024 >>>(v1_d, res_d);

	hipMemcpy(&res_h, res_d, sizeof(int), hipMemcpyDeviceToHost);

	printf("res = %d, expected = %d\n", res_h, N * 3);

	hipFree(v1_d);
	hipFree(res_d);

	hipDeviceSynchronize();
	return (0);
}
