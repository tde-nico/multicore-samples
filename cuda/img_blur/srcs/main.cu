#include "hip/hip_runtime.h"
/**
* Developed By Karan Bhagat
* February 2017
* compile with: nvcc image_stub.cu -o image_stub 
**/

#include "hello.h"

//number of channels i.e. R G B
#define CHANNELS 3
#define BLUR_SIZE 4

unsigned char* loadPPM(const char* path, int* width, int* height);
void writePPM(const char* path, unsigned char* img, int width, int height);

__global__ 	void kernel_blur(unsigned char* d_rgb_image,unsigned char*d_blur_image,
                        int rows,int cols,int bsize) 
{
	int c=threadIdx.x+blockIdx.x*blockDim.x;
	int r=threadIdx.y+blockIdx.y*blockDim.y;
	
    unsigned int red  =0;
    unsigned int green=0;
    unsigned int blue =0;
    int num=0;
	for (int i=-bsize;i<bsize+1;i++)
		for (int j=-bsize;j<bsize+1;j++) {
			if((r+j<0)||(r+j>rows-1)||(c+i<0)||(c+i>cols-1)) continue; 
			red   +=d_rgb_image[3*((r+i)*cols+c+j)];
			green +=d_rgb_image[3*((r+i)*cols+c+j)+1];
			blue  +=d_rgb_image[3*((r+i)*cols+c+j)+2];
			num++;
		}
	
	d_blur_image[3*(c+r*cols)]=(unsigned char)   (red/num);
    d_blur_image[3*(c+r*cols)+1]=(unsigned char) (green/num);
    d_blur_image[3*(c+r*cols)+2]=(unsigned char) (blue/num);



}

int main(int argc, char **argv) 
{
	char* input_file;
	char* output_file;
    int bsize=4;
	//Check for the input file and output file names
	switch(argc) {
		case 4:
			input_file = argv[1];
			output_file = argv[2];
			bsize = atoi(argv[3]);
            break;
		case 3:
			input_file = argv[1];
			output_file = argv[2];
			bsize = 4;
            break;
		default:
			std::cerr << "Usage: <executable> input_file output_file bsize";
			exit(1);
	}
	
	unsigned char *h_rgb_image; //store image's rbg data
	unsigned char *d_rgb_image; //array for storing rgb data on device
	unsigned char *h_blur_image, *d_blur_image; //host and device's blur image data array pointers
	int rows; //number of rows of pixels
	int cols; //number of columns of pixels
	
	//load image into an array and retrieve number of pixels
	h_rgb_image = loadPPM(input_file, &rows, &cols); 

	int total_pixels=rows*cols;
	//allocate memory of host's blur image data array
	h_blur_image = (unsigned char *)malloc(sizeof(unsigned char*) * total_pixels * CHANNELS);

    

	/* ADD YOUR CODE HERE */
	//memcpy(h_blur_image,h_rgb_image,total_pixels*CHANNELS);
	hipMalloc(&d_rgb_image,total_pixels*CHANNELS);
	hipMalloc(&d_blur_image,total_pixels*CHANNELS);
	
	hipMemcpy(d_rgb_image,h_rgb_image,total_pixels*CHANNELS,hipMemcpyHostToDevice);
    dim3 BlockSize(16,16,1);
	dim3 GridSize((cols+15)/16,(15+rows)/16,1);
    	
	kernel_blur<<<GridSize,BlockSize>>>(d_rgb_image,d_blur_image,rows,cols,bsize);
    hipMemcpy(h_blur_image,d_blur_image,total_pixels*CHANNELS,hipMemcpyDeviceToHost);
	
	/* END OF YOUR CODE */

	//output the blurred image
    writePPM(output_file, h_blur_image, rows, cols);
	hipFree(d_rgb_image);
	hipFree(d_blur_image);
	return 0;
}

//function for loading an image into rgb format unsigned char array

unsigned char* loadPPM(const char* path, int* width, int* height) {
	FILE* file = fopen(path, "rb");

	if (!file) {
		fprintf(stderr, "Failed to open file\n");
		return NULL;
	}

	char header[3];
	fscanf(file, "%2s", header);
	if (header[0] != 'P' || header[1] != '6') {
		fprintf(stderr, "Invalid PPM file\n");
		return NULL;
	}

	fscanf(file, "%d %d", width, height);

	int maxColor;
	fscanf(file, "%d", &maxColor);

	fgetc(file);  // Skip single whitespace character

	unsigned char* img = (unsigned char*) malloc((*width) * (*height) * CHANNELS);
	if (!img) {
		fprintf(stderr, "Failed to allocate memory\n");
		return NULL;
	}

	fread(img, CHANNELS, *width * *height, file);

	fclose(file);

	return img;
}

//function for writing blurred image data array to the image file
void writePPM(const char* path, unsigned char* img, int width, int height) {
    FILE* file = fopen(path, "wb");

    if (!file) {
        fprintf(stderr, "Failed to open file\n");
        return;
    }

    fprintf(file, "P6\n%d %d\n255\n", width, height);

    fwrite(img, CHANNELS, width * height, file);

    fclose(file);
}
